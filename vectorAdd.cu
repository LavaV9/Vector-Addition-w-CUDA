#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addVectors(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 8;
    int h_a[N] = {1,2,3,4,5,6,7,8};
    int h_b[N] = {8,7,6,5,4,3,2,1};
    int h_c[N];

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 4;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    addVectors<<<blocks, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
